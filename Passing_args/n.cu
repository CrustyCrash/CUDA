
#include <hip/hip_runtime.h>
#include <iostream>
__global__ void add(int *a, int *b, int *c)
{
    *c = *a + *b;
}
int main(void)
{
    int sum;
    int *device1;
    int *device2;
    int *dev_c;

    hipError_t cudaStatus;

    

   cudaStatus =  hipMalloc((void **)&device1, sizeof(int));
   if(cudaStatus != hipSuccess)
   {
    fprintf(stderr,"Error allocating memory: %s\n",hipGetErrorString(cudaStatus));
    return 1;
   }

    cudaStatus = hipMalloc((void **)&device2, sizeof(int));
    if(cudaStatus != hipSuccess){
        fprintf(stderr,"Error allocating memory: %s\n",hipGetErrorString(cudaStatus));
        hipFree(device1);
        return 1;
    }

    cudaStatus = hipMalloc((void **)&dev_c, sizeof(int));
    if(cudaStatus != hipSuccess){
        fprintf(stderr,"Error allocating memory: %s\n",hipGetErrorString(cudaStatus));
        hipFree(device1);
        hipFree(device2);
        return 1;
    }

    int a = 2;
    int b = 7;

    hipMemcpy(device1, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device2, &b, sizeof(int), hipMemcpyHostToDevice); // Corrected line

    add<<<1, 1>>>(device1, device2, dev_c);

    hipMemcpy(&sum, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(device1);
        hipFree(device2);
        hipFree(dev_c);
        return 1;
    }
    
    printf("2 + 7 = %d\n", sum);
    hipFree(dev_c);
    hipFree(device1);
    hipFree(device2);
    return 0;
}