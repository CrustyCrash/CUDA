#include "hip/hip_runtime.h"
#include <stdio.h>
#include "book.h"

#define N (33 * 1024)
__global__ void sum(int*a , int* b, int* c)
{
    // linearize threadID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid = tid + blockDim.x + gridDim.x; //incrementing by total number of threads in grid
    }
}

int main()
{
    int a[N];
    int b[N];
    int c[N];

    int* dev_a;
    int* dev_b;
    int* dev_c;

    //populating the array on the host
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = -i;
    }

    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    sum <<<128,128>>>(dev_a, dev_b, dev_c);

    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

    //verify that the gpu did the work successfully
    bool verify = true;
    for(int i = 0; i < N; i++)
    {
        if (a[i] + b[i] != c[i])
        {
            printf("Failed at %d + %d != %d\n",a[i],b[i],c[i]);
            verify = false;
        }
    }
    if (verify)
    {
        printf("Program executed successfully!\n");
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}